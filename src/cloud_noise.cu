#include "hip/hip_runtime.h"
#include "cloud_noise.hpp"

#include "glm/gtc/noise.hpp"

#define surface_type float4

__device__ float remap(float domain, float min_x, float max_x, float min_y, float max_y) {
  return (((domain - min_x) / (max_x - min_x)) * (max_y - min_y)) + min_y;
}

__device__ glm::vec3 hash33(glm::vec3 p3) {
  p3 = fract(p3 * glm::vec3(0.1031f, 0.11369f, 0.13787f));
  p3 += dot(p3, glm::vec3(p3.y, p3.x, p3.z) + 19.19f);
  return -1.0f +
         2.0f * glm::fract(glm::vec3((p3.x + p3.y) * p3.z, (p3.x + p3.z) * p3.y,
                                     (p3.y + p3.z) * p3.x));
}

__device__ float worleyNoise(glm::vec3 uv, float freq) {
  glm::vec3 id = glm::floor(uv);
  glm::vec3 p = glm::fract(uv);

  float minDist = 10000.;
  for (float x = -1.; x <= 1.; ++x) {
    for (float y = -1.; y <= 1.; ++y) {
      for (float z = -1.; z <= 1.; ++z) {
        glm::vec3 offset = glm::vec3(x, y, z);
        glm::vec3 h =
            hash33(glm::mod(id + offset, glm::vec3(freq))) * 0.5f + 0.5f;
        h += offset;
        glm::vec3 d = p - h;
        minDist = glm::min(minDist, glm::dot(d, d));
      }
    }
  }

  // inverted worley noise
  return 1. - minDist;
}

__device__ float perlin(const glm::vec3 &pos, float frequency,
                        int octaveCount) {
  const float octaveFrenquencyFactor = 2;

  float sum = 0.0f;
  float weightSum = 0.0f;
  float weight = 0.5f;
  for (int oct = 0; oct < octaveCount; oct++) {
    glm::vec4 p = glm::vec4(pos.x, pos.y, pos.z, 0.0f) * glm::vec4(frequency);
    float val = glm::perlin(p, glm::vec4(frequency));

    sum += val * weight;
    weightSum += weight;

    weight *= weight;
    frequency *= octaveFrenquencyFactor;
  }

  float noise = (sum / weightSum) * 0.5f + 0.5f;
  noise = std::fminf(noise, 1.0f);
  noise = std::fmaxf(noise, 0.0f);
  return noise;
}

__global__ void render(hipSurfaceObject_t surface,
                       CloudsRenderParameters parameters) {
  const int c = blockIdx.x * blockDim.x + threadIdx.x;
  const int r = blockIdx.y * blockDim.y + threadIdx.y;
  if ((c >= parameters.width) || (r >= parameters.height))
    return;

  [[maybe_unused]] const int i = c + r * parameters.width;
  double du = static_cast<double>(c) / static_cast<double>(parameters.width);
  double dv =
      1.0 - (static_cast<double>(r) / static_cast<double>(parameters.height));

  glm::vec4 col = glm::vec4(
      perlin(glm::vec3(parameters.position.x + du, parameters.position.y + dv,
                       parameters.position.z),
             parameters.frequency, parameters.octaves));

  col *= glm::vec4(
      perlin(glm::vec3(parameters.position.x * 0.03151f + du, parameters.position.y * 0.0454f + dv,
                       parameters.position.z),
             parameters.frequency, parameters.octaves));

  col += glm::vec4(
      perlin(glm::vec3(parameters.position.x * 0.6591f + du, parameters.position.y * 0.4564f + dv,
                       parameters.position.z),
             parameters.frequency, parameters.octaves));

  glm::vec4 col2 = glm::vec4(perlin(
      glm::vec3(parameters.position.x * 3.12f + du,
                parameters.position.y * 341.f + dv, parameters.position.z),
      parameters.frequency, parameters.octaves));

  glm::vec4 col3 = glm::vec4(perlin(
      glm::vec3(parameters.position.x * 0.12f + du,
                parameters.position.y * 0.2f + dv, parameters.position.z),
      parameters.frequency, parameters.octaves));

  col += glm::vec4(
      worleyNoise(glm::vec3(parameters.position.x + du,
                            parameters.position.y + dv, parameters.position.z) * parameters.frequency,
                  parameters.frequency));

  col *= 0.25;

  col2 = col;
  col3 = col;

  surface_type output;
  output.x = remap(col.r, parameters.low_cut_l, parameters.high_cut_l, 0.0f, 1.0f);
  output.y = remap(col2.g, parameters.low_cut_m, parameters.high_cut_m, 0.0f, 1.0f);
  output.z = remap(col3.b, parameters.low_cut_h, parameters.high_cut_h, 0.0f, 1.0f);
  // output.y = col.g;
  // output.z = col.b;
  output.w = 1.0f;

  surf2Dwrite(output, surface, c * sizeof(surface_type), r);
}

void generate_cloud_noise(Texture &texture, CloudsRenderParameters &params) {
  const dim3 blockSize(16, 16);
  const dim3 gridSize =
      dim3((texture.get_width() + blockSize.x - 1) / blockSize.x,
           (texture.get_height() + blockSize.y - 1) / blockSize.y);

  hipSurfaceObject_t surface_obj = texture.create_cuda_surface_object();

  render<<<gridSize, blockSize>>>(surface_obj, params);

  texture.destroy_cuda_surface_object(surface_obj);

  hipDeviceSynchronize();
}
