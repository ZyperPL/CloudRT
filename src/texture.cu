#include "texture.hpp"

#include <cassert>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

Texture::Texture(size_t width, size_t height, Format format)
    : width{width}, height{height}, format{format} {
  GLint tex_internal_format = GL_RGB16F;
  GLenum tex_format = GL_RGB;

  switch (format) {
  case Format::Gray:
    tex_internal_format = GL_R32F;
    tex_format = GL_RED;
    break;
  case Format::RGB:
    tex_internal_format = GL_RGB32F;
    tex_format = GL_RGB;
    break;
  case Format::RGBA:
    tex_internal_format = GL_RGBA32F;
    tex_format = GL_RGBA;
    break;
  }

  glEnable(GL_TEXTURE_2D);
  glGenTextures(1, &id);
  glBindTexture(GL_TEXTURE_2D, id);

  glTexImage2D(GL_TEXTURE_2D, 0, tex_internal_format, width, height, 0,
               tex_format, GL_FLOAT, NULL);

  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);

  hipGraphicsGLRegisterImage(&cuda_img_resource, id, GL_TEXTURE_2D,
                              hipGraphicsRegisterFlagsSurfaceLoadStore);
}

Texture::~Texture() {
  hipGraphicsUnregisterResource(cuda_img_resource);
  glDeleteTextures(1, &id);
}

void Texture::update() {
  glBindTexture(GL_TEXTURE_2D, id);
  glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, width, height, GL_RGB, GL_FLOAT,
                  NULL);
}

struct hipGraphicsResource *Texture::get_img_resource() const {
  return cuda_img_resource;
}

void Texture::map_resource(hipArray_t &arr) {
  hipGraphicsMapResources(1, &cuda_img_resource, 0);
  hipGraphicsSubResourceGetMappedArray(&arr, cuda_img_resource, 0, 0);
}

void Texture::unmap_resource() {
  hipGraphicsUnmapResources(1, &cuda_img_resource, 0);
}

hipSurfaceObject_t Texture::create_cuda_surface_object() {
  hipArray_t arr;
  map_resource(arr);

  struct hipResourceDesc desc;
  memset(&desc, 0, sizeof(desc));
  desc.resType = hipResourceTypeArray;
  desc.res.array.array = arr;

  hipSurfaceObject_t obj = 0;
  hipCreateSurfaceObject(&obj, &desc);
  return obj;
}

void Texture::destroy_cuda_surface_object(hipSurfaceObject_t &obj) {
  unmap_resource();
  hipDestroySurfaceObject(obj);
}

hipTextureObject_t Texture::create_cuda_texture_object() {
  hipArray_t arr;
  map_resource(arr);

  struct hipResourceDesc res_desc;
  memset(&res_desc, 0, sizeof(res_desc));
  res_desc.resType = hipResourceTypeArray;
  res_desc.res.array.array = arr;

  hipTextureDesc tex_desc;
  memset(&tex_desc, 0, sizeof(hipTextureDesc));

  tex_desc.normalizedCoords = 1;
  tex_desc.filterMode = hipFilterModeLinear;

  tex_desc.addressMode[0] = hipAddressModeWrap;
  tex_desc.addressMode[1] = hipAddressModeWrap;

  tex_desc.readMode = hipReadModeElementType;

  hipTextureObject_t obj = 0;
  hipCreateTextureObject(&obj, &res_desc, &tex_desc, NULL);
  return obj;
}

void Texture::destroy_cuda_texture_object(hipTextureObject_t &obj) {
  unmap_resource();
  hipDestroyTextureObject(obj);
}

bool Texture::save_to_file(const char *name, FileFormat file_format) {
  const size_t format_bytes = static_cast<size_t>(format);
  const size_t texture_size = format_bytes * width * height;
  float *dest = new float[texture_size];

#if TEXTURE_FILE_RESOURCE_DESC
  struct hipResourceDesc desc;
  memset(&desc, 0, sizeof(desc));
  desc.resType = hipResourceTypeArray;
  cudaGetSurfaceObjectResourceDesc(&desc, cuda_render_surface);
  hipMemcpy2DFromArray(
      dest, 4 * sizeof(float) * out_texture.get_width(), desc.res.array.array,
      0, 0, 4 * sizeof(float) * out_texture.get_width(),
      out_texture.get_height(), hipMemcpyKind::hipMemcpyDeviceToHost);
#endif

  hipArray_t arr;
  map_resource(arr);

  hipError_t ret =
      hipMemcpy2DFromArray(dest, format_bytes * sizeof(float) * width, arr, 0,
                            0, format_bytes * sizeof(float) * width, height,
                            hipMemcpyKind::hipMemcpyDeviceToHost);
  unmap_resource();

  if (ret != hipSuccess)
    return 2;

  unsigned char *data = new unsigned char[texture_size];

  for (size_t i = 0; i < texture_size; i += 4) {
    data[i + 0] = static_cast<unsigned char>(dest[i + 0] * 254.9f);
    data[i + 1] = static_cast<unsigned char>(dest[i + 1] * 254.9f);
    data[i + 2] = static_cast<unsigned char>(dest[i + 2] * 254.9f);
    data[i + 3] = static_cast<unsigned char>(dest[i + 3] * 254.9f);
  }

  printf("Saving file \"%s\" %zux%zu %zu bytes (%zu bytes), %p, format = %d\n",
         name, width, height, format_bytes, texture_size, data, format);

  if (file_format == FileFormat::PNG)
    return stbi_write_png(name, width, height, format_bytes, data,
                          width * format_bytes);

  if (file_format == FileFormat::HDR)
    return stbi_write_hdr(name, width, height, format_bytes, dest);

  return 1;
}
