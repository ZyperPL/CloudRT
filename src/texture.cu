#include "texture.hpp"

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>

Texture::Texture(size_t width, size_t height) : width{width}, height{height} {
  glEnable(GL_TEXTURE_2D);
  glGenTextures(1, &id);
  glBindTexture(GL_TEXTURE_2D, id);

  glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, width, height, 0, GL_RGBA,
               GL_FLOAT, NULL);

  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);

  hipGraphicsGLRegisterImage(&cuda_img_resource, id, GL_TEXTURE_2D,
                              hipGraphicsRegisterFlagsSurfaceLoadStore);
}

Texture::~Texture() {
  hipGraphicsUnregisterResource(cuda_img_resource);
  glDeleteTextures(1, &id);
}

void Texture::update() {
  glBindTexture(GL_TEXTURE_2D, id);
  glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, width, height, GL_RGB, GL_FLOAT,
                  NULL);
}

struct hipGraphicsResource *Texture::get_img_resource() const {
  return cuda_img_resource;
}

void Texture::map_resource(hipArray_t &arr) {
  hipGraphicsMapResources(1, &cuda_img_resource, 0);
  hipGraphicsSubResourceGetMappedArray(&arr, cuda_img_resource, 0, 0);
}

void Texture::unmap_resource() {
  hipGraphicsUnmapResources(1, &cuda_img_resource, 0);
}

hipSurfaceObject_t Texture::create_cuda_surface_object() {
  hipArray_t arr;
  map_resource(arr);

  struct hipResourceDesc desc;
  memset(&desc, 0, sizeof(desc));
  desc.resType = hipResourceTypeArray;
  desc.res.array.array = arr;

  hipSurfaceObject_t obj = 0;
  hipCreateSurfaceObject(&obj, &desc);
  return obj;
}

void Texture::destroy_cuda_surface_object(hipSurfaceObject_t &obj) {
  unmap_resource();
  hipDestroySurfaceObject(obj);
}

hipTextureObject_t Texture::create_cuda_texture_object() {
  hipArray_t arr;
  map_resource(arr);

  struct hipResourceDesc res_desc;
  memset(&res_desc, 0, sizeof(res_desc));
  res_desc.resType = hipResourceTypeArray;
  res_desc.res.array.array = arr;

  hipTextureDesc tex_desc;
  memset(&tex_desc, 0, sizeof(hipTextureDesc));

  tex_desc.normalizedCoords = 1;
  tex_desc.filterMode = hipFilterModeLinear;

  tex_desc.addressMode[0] = hipAddressModeClamp;
  tex_desc.addressMode[1] = hipAddressModeClamp;
  tex_desc.addressMode[2] = hipAddressModeClamp;

  tex_desc.readMode = hipReadModeElementType;

  hipTextureObject_t obj = 0;
  hipCreateTextureObject(&obj, &res_desc, &tex_desc, NULL);
  return obj;
}

void Texture::destroy_cuda_texture_object(hipTextureObject_t &obj) {
  unmap_resource();
  hipDestroyTextureObject(obj);
}
