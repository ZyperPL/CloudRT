#include "hip/hip_runtime.h"
#include "texture.hpp"

const size_t COLOR_COMPONENTS_N = 3;

Texture::Texture(size_t width, size_t height) : width{width}, height{height} {
  glGenBuffers(1, &pbo);
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
  glBufferData(GL_PIXEL_UNPACK_BUFFER,
               width * height * COLOR_COMPONENTS_N * sizeof(GLfloat), 0,
               GL_DYNAMIC_COPY);

  hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, pbo,
                               hipGraphicsRegisterFlagsNone);

  glEnable(GL_TEXTURE_2D);
  glGenTextures(1, &id);
  glBindTexture(GL_TEXTURE_2D, id);

  glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, width, height, 0, GL_RGB, GL_FLOAT,
               NULL);

  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
}

Texture::~Texture() {
  hipGraphicsUnregisterResource(cuda_pbo_resource);
  glDeleteBuffers(1, &pbo);
  glDeleteTextures(1, &id);
}

void Texture::update() {
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
  glBindTexture(GL_TEXTURE_2D, id);
  glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, width, height, GL_RGB, GL_FLOAT,
                  NULL);
}

struct hipGraphicsResource *Texture::get_pbo_resource() const {
  return cuda_pbo_resource;
}

void Texture::map_resource(float3 *&ptr) {
  hipGraphicsMapResources(1, &cuda_pbo_resource, 0);
  hipGraphicsResourceGetMappedPointer((void **)(&ptr), NULL,
                                       cuda_pbo_resource);
}

void Texture::unmap_resource() {
  hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0);
}
